#include "hip/hip_runtime.h"
#include <cudaTypeDef.cuh>

#include <matrixsInfo.h>

namespace sl {
namespace tool {
namespace cudaFunc {
/**
 * @brief               全图像相位高度映射（CUDA加速优化核函数）
 *
 * @param phase         相位图
 * @param rows          行数
 * @param cols          列数
 * @param intrinsic     内参
 * @param coefficient   八参数
 * @param minDepth      最小深度
 * @param maxDepth      最大深度
 * @param depth         深度图
 */
__global__ void phaseHeightMapEigCoe_Device(
    IN const cv::cuda::PtrStep<float> phase, IN const int rows,
    IN const int cols, IN const Eigen::Matrix3f intrinsic,
    IN const Eigen::Vector<float, 8> coefficient, IN const float minDepth,
    IN const float maxDepth, OUT cv::cuda::PtrStep<float> depth) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x > cols - 1 || y > rows - 1)
        return;

    if (phase.ptr(y)[x] == -5.f) {
        depth.ptr(y)[x] = 0.f;
        return;
    }

    Eigen::Matrix3f mapL;
    Eigen::Vector3f mapR, cameraPoint;

    mapL(0, 0) = intrinsic(0, 0);
    mapL(0, 1) = 0;
    mapL(0, 2) = intrinsic(0, 2) - x;
    mapL(1, 0) = 0;
    mapL(1, 1) = intrinsic(1, 1);
    mapL(1, 2) = intrinsic(1, 2) - y;
    mapL(2, 0) = coefficient(0, 0) - coefficient(4, 0) * phase.ptr(y)[x];
    mapL(2, 1) = coefficient(1, 0) - coefficient(5, 0) * phase.ptr(y)[x];
    mapL(2, 2) = coefficient(2, 0) - coefficient(6, 0) * phase.ptr(y)[x];

    mapR(0, 0) = 0;
    mapR(1, 0) = 0;
    mapR(2, 0) = coefficient(7, 0) * phase.ptr(y)[x] - coefficient(3, 0);

    cameraPoint = mapL.inverse() * mapR;
    depth.ptr(y)[x] = cameraPoint.z();
}

/**
 * @brief               全图像相位高度映射（CUDA加速优化核函数）
 *
 * @param depth         深度图
 * @param textureSrc    纹理相机采集的纹理图
 * @param rows          行数
 * @param cols          列数
 * @param intrinsicInvD 深度相机内参矩阵逆矩阵
 * @param intrinsicT    纹理相机内参
 * @param rotateDToT    深度相机到纹理相机的旋转矩阵
 * @param translateDtoT 深度相机到纹理相机的平移矩阵
 * @param textureMapped 映射到深度相机下的纹理
 */
__global__ void reverseMappingTexture_Device(
    IN const cv::cuda::PtrStep<float> depth,
    IN const cv::cuda::PtrStep<uchar3> textureSrc, IN const int rows,
    IN const int cols, IN const Eigen::Matrix3f intrinsicInvD,
    IN const Eigen::Matrix3f intrinsicT, IN const Eigen::Matrix3f rotateDToT,
    IN const Eigen::Vector3f translateDtoT,
    OUT cv::cuda::PtrStep<uchar3> textureMapped) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x > cols - 1 || y > rows - 1)
        return;

    if (depth.ptr(y)[x] == 0.f)
        return;

    Eigen::Vector3f imgPoint(x * depth.ptr(y)[x], y * depth.ptr(y)[x],
                             depth.ptr(y)[x]);
    Eigen::Vector3f texturePoint =
        intrinsicT * (rotateDToT * (intrinsicInvD * imgPoint) + translateDtoT);

    const int xTexture = texturePoint(0, 0) / texturePoint(2, 0);
    const int yTexture = texturePoint(1, 0) / texturePoint(2, 0);

    if (xTexture < 0 || xTexture > cols - 1 || yTexture < 0 ||
        yTexture > rows - 1)
        return;

    textureMapped.ptr(y)[x] = textureSrc.ptr(yTexture)[xTexture];
}
/**
 * @brief               计算纹理图片（CUDA加速优化核函数）
 *
 * @param imgs          纹理合并图
 * @param imgsSize      图片张数
 * @param rows          行数
 * @param cols          列数
 * @param texture       纹理图
 */
__global__ void averageTexture_Device(IN const cv::cuda::PtrStep<uchar> imgs,
                                      IN const int imgsSize, IN const int rows,
                                      IN const int cols,
                                      OUT cv::cuda::PtrStep<uchar> texture) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x > cols - 1 || y > rows - 1)
        return;

    for (size_t i = 0; i < imgsSize; ++i) {
        texture.ptr(y)[3 * x + 0] += imgs.ptr(y)[3 * imgsSize * x + 3 * i + 0];
        texture.ptr(y)[3 * x + 1] += imgs.ptr(y)[3 * imgsSize * x + 3 * i + 1];
        texture.ptr(y)[3 * x + 2] += imgs.ptr(y)[3 * imgsSize * x + 3 * i + 2];
    }
}
/**
 * @brief               过滤相位（CUDA加速优化核函数）
 *
 * @param absPhase      绝对相位
 * @param rows          行数
 * @param cols          列数
 * @param maxTollerance 最大不同量
 * @param kernel        核大小
 * @param out           过滤后的绝对相位图
 */
__global__ void filterPhase_Device(IN const cv::cuda::PtrStep<float> absPhase,
                                   IN const int rows, IN const int cols,
                                   IN const float maxTollerance,
                                   IN const int kernel,
                                   OUT cv::cuda::PtrStep<float> out) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < kernel / 2 || x > cols - kernel / 2 || y < kernel / 2 ||
        y > rows - kernel / 2) {
        return;
    }

    int diffCount = 0;
    const float diffTolerrance = maxTollerance;
    for (int d = -kernel / 2; d < kernel / 2; ++d) {
        for (int k = -kernel / 2; k < kernel / 2; ++k) {
            diffCount = std::abs(absPhase.ptr(y + d)[x + k] -
                                 absPhase.ptr(y)[x]) > diffTolerrance
                            ? diffCount + 1
                            : diffCount;
        }
    }

    out.ptr(y)[x] = diffCount > (kernel / 2 + 1) * (kernel / 2 + 1)
                        ? 0.f
                        : absPhase.ptr(y)[x];
}

void phaseHeightMapEigCoe(
    const cv::cuda::GpuMat &phase, const Eigen::Matrix3f &intrinsic,
    const Eigen::Vector<float, 8> &coefficient, const float minDepth,
    const float maxDepth, cv::cuda::GpuMat &depth,
    const dim3 block,
    cv::cuda::Stream &cvStream) {

    CV_Assert(!phase.empty() && phase.type() == CV_32FC1);

    depth.create(phase.rows, phase.cols, CV_32FC1);
    depth.setTo(0.f);

    const int rows = phase.rows;
    const int cols = phase.cols;

    const dim3 grid((cols + block.x - 1) / block.x,
                    (rows + block.y - 1) / block.y);
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);
    phaseHeightMapEigCoe_Device<<<grid, block, 0, stream>>>(
        phase, phase.rows, phase.cols, intrinsic, coefficient, minDepth,
        maxDepth, depth);
}

void reverseMappingTexture(
    const cv::cuda::GpuMat &depth, const cv::cuda::GpuMat &textureSrc,
    const Eigen::Matrix3f &intrinsicInvD, const Eigen::Matrix3f &intrinsicT,
    const Eigen::Matrix3f &rotateDToT, const Eigen::Vector3f &translateDtoT,
    cv::cuda::GpuMat &textureMapped, const dim3 block,
    cv::cuda::Stream &cvStream) {
    CV_Assert(depth.type() == CV_32FC1 && !depth.empty() &&
              textureSrc.type() == CV_8UC3 && !textureSrc.empty());

    textureMapped.create(textureSrc.size(), CV_8UC3);
    textureMapped.setTo(cv::Scalar(0.f, 0.f, 0.f));

    const int rows = textureSrc.rows;
    const int cols = textureSrc.cols;
    const dim3 grid((cols + block.x - 1) / block.x,
                    (rows + block.y - 1) / block.y);
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);
    reverseMappingTexture_Device<<<grid, block, 0, stream>>>(
        depth, textureSrc, rows, cols, intrinsicInvD, intrinsicT, rotateDToT,
        translateDtoT, textureMapped);
}

void averageTexture(const std::vector<cv::Mat> &imgs, cv::cuda::GpuMat &texture,
                    const dim3 block,
                    cv::cuda::Stream &cvStream) {

    texture.create(imgs[0].size(), CV_8UC3);
    texture.setTo(cv::Scalar(0, 0, 0));

    const int rows = texture.rows;
    const int cols = texture.cols;
    const dim3 grid((cols + block.x - 1) / block.x,
                    (rows + block.y - 1) / block.y);
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);

    cv::Mat mergeImg;
    cv::cuda::GpuMat deviceImgs;
    cv::merge(imgs, mergeImg);
    deviceImgs.upload(mergeImg, cvStream);

    averageTexture_Device<<<grid, block, 0, stream>>>(deviceImgs, imgs.size(),
                                                      rows, cols, texture);
}

void filterPhase(IN const cv::cuda::GpuMat &absPhase, OUT cv::cuda::GpuMat &out,
                 IN const float maxTollerance, IN const int kernel,
                 IN const dim3 block,
                 IN cv::cuda::Stream &cvStream) {
    out.create(absPhase.size(), CV_32FC1);
    out.setTo(0.f);

    const int rows = absPhase.rows;
    const int cols = absPhase.cols;
    const dim3 grid((cols + block.x - 1) / block.x,
                    (rows + block.y - 1) / block.y);
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);

    filterPhase_Device<<<grid, block, 0, stream>>>(absPhase, rows, cols,
                                                   maxTollerance, kernel, out);
}

} // namespace cudaFunc
} // namespace tool
} // namespace sl