#include "hip/hip_runtime.h"
#include <cudaTypeDef.cuh>
#include <hip/hip_runtime_api.h>

#include <phaseSolver.h>
#include <typeDef.h>


namespace sl {
namespace phaseSolver {

namespace cudaFunc {
__global__ void solvePhaseCuda(IN const cv::cuda::PtrStep<uchar> imgs,
                               IN const int imgsSize,
                               IN const int phaseShifteTime, IN const int rows,
                               IN const int cols, IN const float sncThreshold,
                               IN cv::cuda::PtrStep<float> wrapImg,
                               IN cv::cuda::PtrStep<float> conditionImg,
                               IN cv::cuda::PtrStep<float> unwrapImg) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > cols - 1 || x < 0 || y < 0 || y > rows - 1) {
        return;
    }
    // 调制度、相移偏移量、包裹正弦部分、包裹余弦部分
    float snc = 0.f, curShift = 0.f, curSin = 0.f, curCos = 0.f;
    for (size_t i = 0; i < phaseShifteTime; ++i) {
        curShift = i * CV_2PI / phaseShifteTime;
        snc += imgs.ptr(y)[imgsSize * x + i];
        curSin += imgs.ptr(y)[imgsSize * x + i] * sin(curShift);
        curCos += imgs.ptr(y)[imgsSize * x + i] * cos(curShift);
    }

    snc /= phaseShifteTime;
    conditionImg.ptr(y)[x] = snc;

    // 计算包裹相位
    const float wrapVal = -1.f * cuda::std::atan2(curSin, curCos);
    wrapImg.ptr(y)[x] = wrapVal;

    if (snc < sncThreshold) {
        unwrapImg.ptr(y)[x] = 0.f;
        return;
    }
    // 计算计算绝对相位
    int grayCodeK1 = 0, grayCodeK2 = 0, preGrayBit = 0;
    for (size_t i = phaseShifteTime; i < imgsSize; ++i) {
        const int curGrayBit = imgs.ptr(y)[imgsSize * x + i] < snc ? 0 : 1;
        preGrayBit = (i == phaseShifteTime) ? curGrayBit ^ 0 : curGrayBit ^ preGrayBit;
        grayCodeK2 +=
            preGrayBit * cuda::std::pow(2, imgsSize - i - 1);

        if (i != imgsSize - 1) {
            grayCodeK1 += preGrayBit *
                          cuda::std::pow(2, imgsSize - i - 2);
        }
    }

    grayCodeK2 = (grayCodeK2 + 1) / 2;
    if (wrapVal > -CV_PI / 2 && wrapVal < CV_PI / 2) {
        unwrapImg.ptr(y)[x] = wrapVal + CV_2PI * grayCodeK1 + CV_PI;
    }
    else if (wrapVal <= -CV_PI / 2) {
        unwrapImg.ptr(y)[x] = wrapVal + CV_2PI * grayCodeK2 + CV_PI;
    }
    else {
        unwrapImg.ptr(y)[x] = wrapVal + CV_2PI * (grayCodeK2 - 1) + CV_PI;
    }
}

void solvePhase(IN const std::vector<cv::Mat> &imgs,
                OUT PhaseSolverGroupDataDevice &groupData,
                IN const float sncThreshold, IN const int shiftTime,
                IN cv::cuda::Stream &cvStream, IN const dim3 block) {
    CV_Assert(imgs.size() != 0);
    const int rows = imgs[0].rows;
    const int cols = imgs[0].cols;
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    cv::Mat mergeImg;
    cv::cuda::GpuMat deviceImgs;
    cv::merge(imgs, mergeImg);
    deviceImgs.upload(mergeImg, cvStream);

    solvePhaseCuda<<<grid, block, 0, stream>>>(
        deviceImgs, imgs.size(), shiftTime, rows, cols, sncThreshold,
        groupData.__wrapMap, groupData.__textureMap, groupData.__unwrapMap);
}
} // namespace cudaFunc
} // namespace phaseSolver
} // namespace sl