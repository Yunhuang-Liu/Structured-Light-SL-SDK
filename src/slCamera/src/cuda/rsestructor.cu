#include "hip/hip_runtime.h"
#include <cudaTypeDef.cuh>

#include <atomic>

#include <restructor.h>

namespace sl {
namespace restructor {

namespace cudaFunc {
__global__ void matchAndTriangulateCUDA(
    cv::cuda::PtrStep<float> leftImg, cv::cuda::PtrStep<float> rightImg,
    const int rows, const int cols, const int minDisparity,
    const int maxDisparity, const float minDepth, const float maxDepth,
    const float maximumCost, const Eigen::Matrix4f Q, const Eigen::Matrix3f M1,
    const Eigen::Matrix3f M3, const Eigen::Matrix3f R, const Eigen::Vector3f T,
    const Eigen::Matrix3f R1_inv, cv::cuda::PtrStep<float> mapDepth,
    const bool isMapToPreAxes, const bool isMapToColorCamera) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < cols && y < rows) {
        if (0 >= leftImg.ptr(y)[x]) {
            return;
        }
        const float f = Q(2, 3);
        const float tx = -1.0 / Q(3, 2);
        const float cxlr = Q(3, 3) * tx;
        const float cx = -1.0 * Q(0, 3);
        const float cy = -1.0 * Q(1, 3);
        float cost = 0;
        int k = 0;
        float minCost = FLT_MAX;
        bool sucessFind = false;
        for (int d = minDisparity; d < maxDisparity; ++d) {
            if (x - d < 0 || x - d > cols - 1) {
                continue;
            }

            cost = std::abs(leftImg.ptr(y)[x] - rightImg.ptr(y)[x - d]);

            if(sucessFind) {
                if(cost > minCost) {
                    break;
                }
            }

            if (cost < minCost) {
                minCost = cost;
                k = d;
                sucessFind = minCost < maximumCost ? true : false;
            }
        }

        if(!sucessFind) {
            return;
        }

        float dived = rightImg.ptr(y)[x - k + 1] - rightImg.ptr(y)[x - k - 1];

        if (std::abs(dived) < 0.001) {
            dived = 0.001;
        }

        float disparity =
            k + 2 * (rightImg.ptr(y)[x - k] - leftImg.ptr(y)[x]) / dived;

        if (disparity < minDisparity || disparity > maxDisparity || std::abs(disparity - k) > 1.f) {
            return;
        }

        Eigen::Vector3f vertex;
        vertex(0, 0) = -1.0f * tx * (x - cx) / (disparity - cxlr);
        vertex(1, 0) = -1.0f * tx * (y - cy) / (disparity - cxlr);
        vertex(2, 0) = -1.0f * tx * f / (disparity - cxlr);

        const Eigen::Vector3f depthVertex =
            isMapToPreAxes ? R1_inv * vertex : vertex;
        const Eigen::Vector3f colorVertex =
            isMapToColorCamera ? R * (R1_inv * vertex) + T : depthVertex;

        if (isMapToPreAxes) {
            const Eigen::Vector3f imgMapped =
                isMapToColorCamera ? M3 * colorVertex : M1 * colorVertex;
            const int x_maped = imgMapped(0, 0) / imgMapped(2, 0);
            const int y_maped = imgMapped(1, 0) / imgMapped(2, 0);
            const float depthMaped = colorVertex(2, 0);
            if (x_maped < cols && y_maped < rows && 0 <= x_maped &&
                0 <= y_maped) {
                if (depthMaped < minDepth || depthMaped > maxDepth) {
                    atomicExch(&mapDepth.ptr(y_maped)[x_maped], 0);
                }
                else {
                    atomicExch(&mapDepth.ptr(y_maped)[x_maped], depthMaped);
                }
            }
        } 
        else {
            mapDepth.ptr(y)[x] = colorVertex(2, 0);
        }
    }
}

void getDepthMap(const cv::cuda::GpuMat &leftImg,
                 const cv::cuda::GpuMat &rightImg,
                 const RestructParamater param, const Eigen::Matrix4f &Q,
                 const Eigen::Matrix3f &M1, const Eigen::Matrix3f &M3,
                 const Eigen::Matrix3f &R, const Eigen::Vector3f &T,
                 const Eigen::Matrix3f &R1Inv, cv::cuda::GpuMat &depthMap,
                 cv::cuda::Stream &cvStream, const dim3 block) {
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);
    dim3 grid((leftImg.cols + block.x - 1) / block.x,
              (leftImg.rows + block.y - 1) / block.y, 1);
    matchAndTriangulateCUDA<<<grid, block, 0, stream>>>(
        leftImg, rightImg, leftImg.rows, leftImg.cols, param.__minDisparity,
        param.__maxDisparity, param.__minDepth, param.__maxDepth,
        param.__maximumCost, Q, M1, M3, R, T, R1Inv, depthMap,
        param.__isMapToPreDepthAxes, param.__isMapToColorCamera);
}
} // namespace cudaFunc
} // namespace restructor
} // namespace sl
