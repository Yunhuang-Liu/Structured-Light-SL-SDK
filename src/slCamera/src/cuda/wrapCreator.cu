#include "hip/hip_runtime.h"
#include <cudaTypeDef.cuh>

#include <typeDef.h>

namespace sl {
namespace wrapCreator {
namespace cudaFunc {
__global__ void solveWrapCuda(IN const cv::cuda::PtrStep<uchar> imgs,
                              IN const int imgsSize, IN const int rows,
                              IN const int cols, 
                              IN cv::cuda::PtrStep<float> wrapImg,
                              IN cv::cuda::PtrStep<float> conditionImg) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > cols - 1 || x < 0 || y < 0 || y > rows - 1) {
        return;
    }
    // 调制度、相移偏移量、包裹正弦部分、包裹余弦部分
    float snc = 0.f, curShift = 0.f, curSin = 0.f, curCos = 0.f;
    for (size_t i = 0; i < imgsSize; ++i) {
        curShift = i * CV_2PI / imgsSize;
        snc += imgs.ptr(y)[imgsSize * x + i];
        curSin += imgs.ptr(y)[imgsSize * x + i] * sin(curShift);
        curCos += imgs.ptr(y)[imgsSize * x + i] * cos(curShift);
    }
    snc /= imgsSize;

    // 计算包裹相位
    const float wrapVal = -1.f * cuda::std::atan2(curSin, curCos);
    
    wrapImg.ptr(y)[x] = wrapVal;
    conditionImg.ptr(y)[x] = snc;
}

void getWrapImgSync(IN const std::vector<cv::Mat> &imgs,
                    OUT cv::cuda::GpuMat &wrapImg, OUT cv::cuda::GpuMat &conditionImg,
                    IN cv::cuda::Stream &cvStream, IN const dim3 block) {
    CV_Assert(imgs.size() != 0);
    const int rows = imgs[0].rows;
    const int cols = imgs[0].cols;
    hipStream_t stream = cv::cuda::StreamAccessor::getStream(cvStream);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    cv::Mat mergeImg;
    cv::cuda::GpuMat deviceImgs;
    cv::merge(imgs, mergeImg);
    deviceImgs.upload(mergeImg, cvStream);

    solveWrapCuda<<<grid, block, 0, stream>>>(
        deviceImgs, imgs.size(), rows, cols,
        wrapImg, conditionImg);
}
} // namespace cudaFunc
} // namespace wrapCreator
} // namespace sl